//GPULookupsPerf.cu

#include <stdio.h>
#include <assert.h>
#include <limits.h>
#include <hip/hip_runtime_api.h>

#include "../../mt19937ar.h"
#include "quotientFilter.cuh"

#ifndef NOT_FOUND
#define NOT_FOUND UINT_MAX
#endif

void generateRandomNumbers(unsigned int *numberArray, unsigned int n)
{
    for (int i = 0; i < n; i++){
        numberArray[i] = genrand_int32();
    }
}

void CUDAErrorCheck()
{
    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

int main(int argc, char* argv[])
{
    assert(argc == 4);
    int q = atoi(argv[1]);
    int r = atoi(argv[2]);
    float alpha = atof(argv[3]);

    //Initialize filter
    struct quotient_filter d_qfilter;
    initFilterGPU(&d_qfilter, q, r);
    hipMemset(d_qfilter.table, 0, calcNumSlotsGPU(q, r) * sizeof(unsigned char));
    
    //Generate set of random numbers
    int numValues = alpha * (1 << q);
    unsigned int* h_randomValues = new unsigned int[numValues];
    init_genrand(time(NULL));   //initialize random number generator
    generateRandomNumbers(h_randomValues, numValues);
    unsigned int* d_randomValues;
    hipMalloc((void**) &d_randomValues, numValues * sizeof(unsigned int));
    hipMemcpy(d_randomValues, h_randomValues, numValues * sizeof(unsigned int), hipMemcpyHostToDevice);

//Random Inserts
    CUDAErrorCheck();
    float filterBuildTime = bulkBuildSegmentedLayouts(d_qfilter, numValues, d_randomValues, true);
    CUDAErrorCheck();
//    printf("Insert rate = %f million ops/sec\n", numValues / filterBuildTime / 1000);

//Successful lookups
    //Use inserted values as lookup values
    unsigned int* d_successfulLookupValues;
    hipMalloc((void**) &d_successfulLookupValues, numValues * sizeof(int));
    hipMemcpy(d_successfulLookupValues, h_randomValues, numValues * sizeof(int), hipMemcpyHostToDevice);

    //Output array
    unsigned int* d_returnValues;
    hipMalloc((void**) &d_returnValues, numValues * sizeof(unsigned int));
    hipMemset(&d_returnValues, 0, numValues * sizeof(unsigned int));
    
    CUDAErrorCheck();
    //float lookupTime = launchSortedLookups(d_qfilter, numValues, d_successfulLookupValues, d_returnValues);
    float lookupTime = launchUnsortedLookups(d_qfilter, numValues, d_successfulLookupValues, d_returnValues);
    CUDAErrorCheck();

    printf("Lookup rate = %f million ops/sec\n", numValues / lookupTime / 1000);
    //printf("%f\n", numValues / lookupTime / 1000);

    //Free Memory
    hipFree(d_qfilter.table);
    delete[] h_randomValues;
    hipFree(d_randomValues);
    hipFree(d_successfulLookupValues);
    hipFree(d_returnValues);
    hipDeviceReset();

    return 0;
}
