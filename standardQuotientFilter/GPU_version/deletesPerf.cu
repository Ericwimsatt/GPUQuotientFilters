//deletesPerf.cu

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>

#include "../../mt19937ar.h"
#include "quotientFilter.cuh"

#ifndef LOW_BIT_MASK
#define LOW_BIT_MASK(n) ((1U << n) - 1U)
#endif

void generateRandomNumbers(unsigned int *numberArray, int n)
{
    for (int i = 0; i < n; i++){
        numberArray[i] = genrand_int32();
    }
}

int main(int argc, char* argv[])
{
    assert(argc == 5);
    int q = atoi(argv[1]);
    int r = atoi(argv[2]);
    float alpha = atof(argv[3]);    //initial fill %
    int batchSize = atoi(argv[4]);  //size of batch to delete after build

    //Initialize filter
    struct quotient_filter d_qfilter;
    initFilterGPU(&d_qfilter, q, r);
    hipMemset(d_qfilter.table, 0, calcNumSlotsGPU(q, r) * sizeof(unsigned char));

    //Generate set of random numbers
    int numValues = alpha * (1 << q);
    unsigned int* h_randomValues = new unsigned int[numValues];
    init_genrand(time(NULL));   //initialize random number generator
    generateRandomNumbers(h_randomValues, numValues);
    unsigned int* d_randomValues;
    hipMalloc((void**) &d_randomValues, numValues * sizeof(unsigned int));
    hipMemcpy(d_randomValues, h_randomValues, numValues * sizeof(unsigned int), hipMemcpyHostToDevice);

//Random Inserts
    float filterBuildTime = bulkBuildSegmentedLayouts(d_qfilter, numValues, d_randomValues, true);

//Delete Some of the Items
    unsigned int* d_deleteValues;
    hipMalloc((void**) &d_deleteValues, batchSize * sizeof(unsigned int));
    hipMemcpy(d_deleteValues, h_randomValues, batchSize * sizeof(unsigned int), hipMemcpyHostToDevice);

    //Delete kernel
    float deleteTime = superclusterDeletes(d_qfilter, batchSize, d_deleteValues);
    printf("Delete rate = %f million ops/sec\n", batchSize / deleteTime / 1000);
    //printf("%f\n", batchSize / deleteTime / 1000);

    //Free Memory
    hipFree(d_qfilter.table);
    hipFree(d_randomValues);
    delete[] h_randomValues;
    hipFree(d_deleteValues);
    hipDeviceReset();

    return 0;
}
