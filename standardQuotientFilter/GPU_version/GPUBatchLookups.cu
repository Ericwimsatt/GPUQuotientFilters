//GPUBatchLookups.cu

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>

#include "../../mt19937ar.h"
#include "quotientFilter.cuh"

#ifndef NOT_FOUND
#define NOT_FOUND UINT_MAX
#endif

void generateRandomNumbers(unsigned int *numberArray, unsigned int n)
{
    for (int i = 0; i < n; i++){
        numberArray[i] = genrand_int32();
    }
}

void CUDAErrorCheck()
{
    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    errSync = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

int main(int argc, char* argv[])
{
    assert(argc == 5);
    int q = atoi(argv[1]);
    int r = atoi(argv[2]);
    float alpha = atof(argv[3]);    //initial fill %
    int batchSize = atoi(argv[4]);  //size of batch to insert after build

    //Initialize filter
    struct quotient_filter d_qfilter;
    initFilterGPU(&d_qfilter, q, r);
    hipMemset(d_qfilter.table, 0, calcNumSlotsGPU(q, r) * sizeof(unsigned char));
    
    //Generate set of random numbers
    int numValues = alpha * (1 << q);
    unsigned int* h_randomValues = new unsigned int[numValues];
    init_genrand(time(NULL));   //initialize random number generator
    generateRandomNumbers(h_randomValues, numValues);
    unsigned int* d_randomValues;
    hipMalloc((void**) &d_randomValues, numValues * sizeof(unsigned int));
    hipMemcpy(d_randomValues, h_randomValues, numValues * sizeof(unsigned int), hipMemcpyHostToDevice);

//Random Inserts
    CUDAErrorCheck();
    float filterBuildTime = bulkBuildSegmentedLayouts(d_qfilter, numValues, d_randomValues, true);
    CUDAErrorCheck();
//    printf("Insert rate = %f million ops/sec\n", numValues / filterBuildTime / 1000);

//New random batch lookups
    //Generate values for random lookups
    unsigned int* h_batchLookupValues = new unsigned int[batchSize];
    generateRandomNumbers(h_batchLookupValues, batchSize);

    //Array of lookup values
    unsigned int* d_batchLookupValues;
    hipMalloc((void**) &d_batchLookupValues, batchSize * sizeof(int));
    hipMemcpy(d_batchLookupValues, h_batchLookupValues, batchSize * sizeof(int), hipMemcpyHostToDevice);

    //Output array
    unsigned int* d_batchReturnValues;
    hipMalloc((void**) &d_batchReturnValues, batchSize * sizeof(unsigned int));
    hipMemset(&d_batchReturnValues, 0, batchSize * sizeof(unsigned int));
    
    CUDAErrorCheck();
    //float batchLookupTime = launchSortedLookups(d_qfilter, batchSize, d_batchLookupValues, d_batchReturnValues);
    float batchLookupTime = launchUnsortedLookups(d_qfilter, batchSize, d_batchLookupValues, d_batchReturnValues);
    CUDAErrorCheck();
    printf("Batch lookup rate = %f million ops/sec\n", batchSize / batchLookupTime / 1000);
    //printf("%f\n", batchSize / batchLookupTime / 1000);

    //Free Memory
    hipFree(d_qfilter.table);
    delete[] h_randomValues;
    hipFree(d_randomValues);
    delete[] h_batchLookupValues;
    hipFree(d_batchLookupValues);
    hipFree(d_batchReturnValues);
    hipDeviceReset();

    return 0;
}
