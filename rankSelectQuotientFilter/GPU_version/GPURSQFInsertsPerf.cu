//GPURSQFInsertsPerf.cu

#include <stdio.h>
#include <assert.h>

#include "../../mt19937ar.h"
#include "RSQF.cuh"

void generateRandomNumbers(unsigned int *numberArray, unsigned int n)
{
    for (int i = 0; i < n; i++){
        numberArray[i] = genrand_int32();
    }   
}

int main(int argc, char* argv[])
{
    assert(argc == 3); 
    int q = atoi(argv[1]);
    float alpha = atof(argv[2]);

//Initialize the filter:
    struct countingQuotientFilterGPU test_cqf_gpu;
    initCQFGPU(&test_cqf_gpu, q); 

    //Generate random numbers:
    unsigned int numValues = alpha * (1 << q); 
    unsigned int* h_randomValues = new unsigned int[numValues];
    init_genrand(time(NULL));       //initialize random number generator
    generateRandomNumbers(h_randomValues, numValues);
    unsigned int* d_randomValues;
    hipMalloc((void**) &d_randomValues, numValues * sizeof(unsigned int));
    hipMemcpy(d_randomValues, h_randomValues, numValues * sizeof(unsigned int), hipMemcpyHostToDevice);

//Inserts
    //Allocate output array
    int* d_insertReturnValues;
    hipMalloc((void**) &d_insertReturnValues, numValues * sizeof(int));

    //Insert kernel
    float insertTime = insertGPU(test_cqf_gpu, numValues, d_randomValues, d_insertReturnValues);
    printf("%f\n", numValues / insertTime / 1000);

    //Free Memory
    hipFree(test_cqf_gpu.blocks);
    delete[] h_randomValues;
    hipFree(d_randomValues);
    hipFree(d_insertReturnValues);
    hipDeviceReset();

    return 0;
}
